#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */
//  /usr/local/cuda-10.2/samples/1_Utilities/deviceQuery
// 2 Muliprocesadores
// 128 Cores por mp
// 256 Cores
// Maximum number of threads per multiprocessor:  2048
// Maximum number of threads per block:           1024
// Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
// Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)

#define TxB 1024

__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  // The mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  // Since it does not matter the relative position of a pixel
  // the block - grid assign strategy will simply be to cover
  // all pixels secuencially in 'x' axis
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer -- numero de bloques
  // tamaño bloque
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}